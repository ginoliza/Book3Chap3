#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "lodepng.h"

#define CHECK_ERROR(call) { 
	hipError_t err = call; 
	if (err != hipSuccess) { 
		printf("%s en %s , linea %d\n", hipGetErrorString(err), __FILE__, __LINE__); 
		exit(err); 
	} 
}

#define CHANNELS 4

__global__
void colorToGrayscaleConversionKernel(unsigned char *Pin, unsigned char *Pout, int width, int height) {
    int Col = blockDim.x * blockIdx.x + threadIdx.x;
    int Row = blockDim.y * blockIdx.y + threadIdx.y;
        
    if ( Col < width  && Row < height) {
                
        int greyOffset = Row * width + Col;
               
        int rgbOffset = greyOffset * CHANNELS;
        unsigned char r = Pin[rgbOffset ];		
        unsigned char g = Pin[rgbOffset + 1];	
        unsigned char b = Pin[rgbOffset + 2];	
                
        Pout[rgbOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
        Pout[rgbOffset+1] = 0.21f*r + 0.71f*g + 0.07f*b;
        Pout[rgbOffset+2] = 0.21f*r + 0.71f*g + 0.07f*b;
        Pout[rgbOffset+3] = 255;
    }
}


void colorToGrayscaleConversion(unsigned char *h_Pin, unsigned char *h_Pout, int m, int n) {
    
    int size = (m*n*4)*sizeof(unsigned char);
    unsigned char *d_Pin, *d_Pout;

    CHECK_ERROR(hipMalloc((void**)&d_Pin, size));
    CHECK_ERROR(hipMalloc((void**)&d_Pout, size));

    hipMemcpy(d_Pin, h_Pin, size, hipMemcpyHostToDevice);

    dim3 dimGrid(ceil(m / 16.0),ceil(n / 16.0),1);
    dim3 dimBlock(16, 16,1);
    colorToGrayscaleConversionKernel<<<dimGrid, dimBlock>>>(d_Pin, d_Pout, m, n);

    hipMemcpy(h_Pout, d_Pout, size, hipMemcpyDeviceToHost);

    hipFree(d_Pin);
    hipFree(d_Pout);
}

unsigned char* decodeOneStep(const char* filename)
{
    unsigned error;
    unsigned char* image;
    unsigned width, height;
    
    error = lodepng_decode32_file(&image, &width, &height, filename);
    if(error) printf("error %u: %s\n", error, lodepng_error_text(error));
    
    return image;
}

void encodeOneStep(const char* filename, unsigned char* image, int width, int height)
{
    unsigned error = lodepng_encode32_file(filename, image, width, height);
 
    if(error) printf("error %u: %s\n", error, lodepng_error_text(error));
}

int main(int argc, char *argv[]) {
 
    if (argc != 2) {        
        exit(1);
    }
    const char *filename = argv[1];
    
    unsigned char *h_Pin, *h_Pout;
    
    int m = 512; 
    int n = 512; 
    
    h_Pin = (unsigned char*)malloc(sizeof(unsigned char)*(n*m));
    h_Pout = (unsigned char*)malloc(sizeof(unsigned char)*(n*m*4));
    
    h_Pin = decodeOneStep(filename);
    
    colorToGrayscaleConversion(h_Pin, h_Pout, m, n);    
    
    encodeOneStep("image_converted.png", h_Pout, m, n);
    
    free(h_Pin);
    free(h_Pout);
    
    return 0;
}